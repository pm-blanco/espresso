#include "hip/hip_runtime.h"
//======================================================================================================================
//
//  This file is part of waLBerla. waLBerla is free software: you can
//  redistribute it and/or modify it under the terms of the GNU General Public
//  License as published by the Free Software Foundation, either version 3 of
//  the License, or (at your option) any later version.
//
//  waLBerla is distributed in the hope that it will be useful, but WITHOUT
//  ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or
//  FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public License
//  for more details.
//
//  You should have received a copy of the GNU General Public License along
//  with waLBerla (see COPYING.txt). If not, see <http://www.gnu.org/licenses/>.
//
//! \\file DynamicUBBSinglePrecisionCUDA.cpp
//! \\author pystencils
//======================================================================================================================

// kernel generated with pystencils v1.3.7, lbmpy v1.3.7, sympy v1.12.1, lbmpy_walberla/pystencils_walberla from waLBerla commit f36fa0a68bae59f0b516f6587ea8fa7c24a41141

#include "DynamicUBBSinglePrecisionCUDA.h"
#include "core/DataTypes.h"
#include "core/Macros.h"
#include "gpu/ErrorChecking.h"

#define FUNC_PREFIX __global__

using namespace std;

namespace walberla {
namespace lbm {

#if defined(__NVCC__)
#define RESTRICT __restrict__
#if defined(__NVCC_DIAG_PRAGMA_SUPPORT__)
#pragma nv_diagnostic push
#pragma nv_diag_suppress 177 // unused variable
#else
#pragma push
#pragma diag_suppress 177 // unused variable
#endif                    // defined(__NVCC_DIAG_PRAGMA_SUPPORT__)
#elif defined(__clang__)
#if defined(__CUDA__)
#if defined(__CUDA_ARCH__)
// clang compiling CUDA code in device mode
#define RESTRICT __restrict__
#pragma clang diagnostic push
#pragma clang diagnostic ignored "-Wstrict-aliasing"
#pragma clang diagnostic ignored "-Wunused-variable"
#pragma clang diagnostic ignored "-Wconversion"
#pragma clang diagnostic ignored "-Wsign-compare"
#else
// clang compiling CUDA code in host mode
#define RESTRICT __restrict__
#pragma clang diagnostic push
#pragma clang diagnostic ignored "-Wstrict-aliasing"
#pragma clang diagnostic ignored "-Wunused-variable"
#pragma clang diagnostic ignored "-Wconversion"
#pragma clang diagnostic ignored "-Wsign-compare"
#endif // defined(__CUDA_ARCH__)
#endif // defined(__CUDA__)
#elif defined(__GNUC__) or defined(__GNUG__)
#define RESTRICT __restrict__
#pragma GCC diagnostic push
#pragma GCC diagnostic ignored "-Wstrict-aliasing"
#pragma GCC diagnostic ignored "-Wunused-variable"
#pragma GCC diagnostic ignored "-Wconversion"
#elif defined(_MSC_VER)
#define RESTRICT __restrict
#else
#define RESTRICT
#endif

// NOLINTBEGIN(readability-non-const-parameter*)
namespace internal_dynamicubbsingleprecisioncuda_boundary_DynamicUBBSinglePrecisionCUDA {
static FUNC_PREFIX __launch_bounds__(256) void dynamicubbsingleprecisioncuda_boundary_DynamicUBBSinglePrecisionCUDA(uint8_t *RESTRICT const _data_indexVector, float *RESTRICT _data_pdfs, int64_t const _stride_pdfs_0, int64_t const _stride_pdfs_1, int64_t const _stride_pdfs_2, int64_t const _stride_pdfs_3, int32_t indexVectorSize) {

  const int32_t f_in_inv_dir_idx[] = {0, 2, 1, 4, 3, 6, 5, 10, 9, 8, 7, 16, 15, 18, 17, 12, 11, 14, 13};

  const float weights[] = {((float)(0.33333333333333333)), ((float)(0.055555555555555556)), ((float)(0.055555555555555556)), ((float)(0.055555555555555556)), ((float)(0.055555555555555556)), ((float)(0.055555555555555556)), ((float)(0.055555555555555556)), ((float)(0.027777777777777778)), ((float)(0.027777777777777778)), ((float)(0.027777777777777778)), ((float)(0.027777777777777778)), ((float)(0.027777777777777778)), ((float)(0.027777777777777778)), ((float)(0.027777777777777778)), ((float)(0.027777777777777778)), ((float)(0.027777777777777778)), ((float)(0.027777777777777778)), ((float)(0.027777777777777778)), ((float)(0.027777777777777778))};

  const int32_t neighbour_offset_x[] = {0, 0, 0, -1, 1, 0, 0, -1, 1, -1, 1, 0, 0, -1, 1, 0, 0, -1, 1};
  const int32_t neighbour_offset_y[] = {0, 1, -1, 0, 0, 0, 0, 1, 1, -1, -1, 1, -1, 0, 0, 1, -1, 0, 0};
  const int32_t neighbour_offset_z[] = {0, 0, 0, 0, 0, 1, -1, 0, 0, 0, 0, 1, 1, 1, 1, -1, -1, -1, -1};

  if (blockDim.x * blockIdx.x + threadIdx.x < indexVectorSize) {
    uint8_t *RESTRICT _data_indexVector_10 = _data_indexVector;
    const int32_t x = *((int32_t *)(&_data_indexVector_10[28 * blockDim.x * blockIdx.x + 28 * threadIdx.x]));
    uint8_t *RESTRICT _data_indexVector_14 = _data_indexVector + 4;
    const int32_t y = *((int32_t *)(&_data_indexVector_14[28 * blockDim.x * blockIdx.x + 28 * threadIdx.x]));
    uint8_t *RESTRICT _data_indexVector_18 = _data_indexVector + 8;
    const int32_t z = *((int32_t *)(&_data_indexVector_18[28 * blockDim.x * blockIdx.x + 28 * threadIdx.x]));
    uint8_t *RESTRICT _data_indexVector_112 = _data_indexVector + 12;
    const int32_t dir = *((int32_t *)(&_data_indexVector_112[28 * blockDim.x * blockIdx.x + 28 * threadIdx.x]));
    float *RESTRICT _data_pdfs_10_2m1_318 = _data_pdfs + _stride_pdfs_1 * y + _stride_pdfs_2 * z - _stride_pdfs_2 + 18 * _stride_pdfs_3;
    float *RESTRICT _data_pdfs_10_20_34 = _data_pdfs + _stride_pdfs_1 * y + _stride_pdfs_2 * z + 4 * _stride_pdfs_3;
    float *RESTRICT _data_pdfs_11_20_38 = _data_pdfs + _stride_pdfs_1 * y + _stride_pdfs_1 + _stride_pdfs_2 * z + 8 * _stride_pdfs_3;
    float *RESTRICT _data_pdfs_1m1_20_310 = _data_pdfs + _stride_pdfs_1 * y - _stride_pdfs_1 + _stride_pdfs_2 * z + 10 * _stride_pdfs_3;
    float *RESTRICT _data_pdfs_10_21_314 = _data_pdfs + _stride_pdfs_1 * y + _stride_pdfs_2 * z + _stride_pdfs_2 + 14 * _stride_pdfs_3;
    const float vel0Term = _data_pdfs_10_20_34[_stride_pdfs_0 * x + _stride_pdfs_0] + _data_pdfs_10_21_314[_stride_pdfs_0 * x + _stride_pdfs_0] + _data_pdfs_10_2m1_318[_stride_pdfs_0 * x + _stride_pdfs_0] + _data_pdfs_11_20_38[_stride_pdfs_0 * x + _stride_pdfs_0] + _data_pdfs_1m1_20_310[_stride_pdfs_0 * x + _stride_pdfs_0];
    float *RESTRICT _data_pdfs_11_2m1_315 = _data_pdfs + _stride_pdfs_1 * y + _stride_pdfs_1 + _stride_pdfs_2 * z - _stride_pdfs_2 + 15 * _stride_pdfs_3;
    float *RESTRICT _data_pdfs_11_20_37 = _data_pdfs + _stride_pdfs_1 * y + _stride_pdfs_1 + _stride_pdfs_2 * z + 7 * _stride_pdfs_3;
    float *RESTRICT _data_pdfs_11_20_31 = _data_pdfs + _stride_pdfs_1 * y + _stride_pdfs_1 + _stride_pdfs_2 * z + _stride_pdfs_3;
    float *RESTRICT _data_pdfs_11_21_311 = _data_pdfs + _stride_pdfs_1 * y + _stride_pdfs_1 + _stride_pdfs_2 * z + _stride_pdfs_2 + 11 * _stride_pdfs_3;
    const float vel1Term = _data_pdfs_11_20_31[_stride_pdfs_0 * x] + _data_pdfs_11_20_37[_stride_pdfs_0 * x - _stride_pdfs_0] + _data_pdfs_11_21_311[_stride_pdfs_0 * x] + _data_pdfs_11_2m1_315[_stride_pdfs_0 * x];
    float *RESTRICT _data_pdfs_1m1_21_312 = _data_pdfs + _stride_pdfs_1 * y - _stride_pdfs_1 + _stride_pdfs_2 * z + _stride_pdfs_2 + 12 * _stride_pdfs_3;
    float *RESTRICT _data_pdfs_10_21_313 = _data_pdfs + _stride_pdfs_1 * y + _stride_pdfs_2 * z + _stride_pdfs_2 + 13 * _stride_pdfs_3;
    float *RESTRICT _data_pdfs_10_21_35 = _data_pdfs + _stride_pdfs_1 * y + _stride_pdfs_2 * z + _stride_pdfs_2 + 5 * _stride_pdfs_3;
    const float vel2Term = _data_pdfs_10_21_313[_stride_pdfs_0 * x - _stride_pdfs_0] + _data_pdfs_10_21_35[_stride_pdfs_0 * x] + _data_pdfs_1m1_21_312[_stride_pdfs_0 * x];
    float *RESTRICT _data_pdfs_1m1_2m1_316 = _data_pdfs + _stride_pdfs_1 * y - _stride_pdfs_1 + _stride_pdfs_2 * z - _stride_pdfs_2 + 16 * _stride_pdfs_3;
    float *RESTRICT _data_pdfs_10_2m1_317 = _data_pdfs + _stride_pdfs_1 * y + _stride_pdfs_2 * z - _stride_pdfs_2 + 17 * _stride_pdfs_3;
    float *RESTRICT _data_pdfs_10_2m1_36 = _data_pdfs + _stride_pdfs_1 * y + _stride_pdfs_2 * z - _stride_pdfs_2 + 6 * _stride_pdfs_3;
    float *RESTRICT _data_pdfs_10_20_30 = _data_pdfs + _stride_pdfs_1 * y + _stride_pdfs_2 * z;
    float *RESTRICT _data_pdfs_1m1_20_39 = _data_pdfs + _stride_pdfs_1 * y - _stride_pdfs_1 + _stride_pdfs_2 * z + 9 * _stride_pdfs_3;
    float *RESTRICT _data_pdfs_1m1_20_32 = _data_pdfs + _stride_pdfs_1 * y - _stride_pdfs_1 + _stride_pdfs_2 * z + 2 * _stride_pdfs_3;
    float *RESTRICT _data_pdfs_10_20_33 = _data_pdfs + _stride_pdfs_1 * y + _stride_pdfs_2 * z + 3 * _stride_pdfs_3;
    const float rho = vel0Term + vel1Term + vel2Term + _data_pdfs_10_20_30[_stride_pdfs_0 * x] + _data_pdfs_10_20_33[_stride_pdfs_0 * x - _stride_pdfs_0] + _data_pdfs_10_2m1_317[_stride_pdfs_0 * x - _stride_pdfs_0] + _data_pdfs_10_2m1_36[_stride_pdfs_0 * x] + _data_pdfs_1m1_20_32[_stride_pdfs_0 * x] + _data_pdfs_1m1_20_39[_stride_pdfs_0 * x - _stride_pdfs_0] + _data_pdfs_1m1_2m1_316[_stride_pdfs_0 * x];
    float *RESTRICT _data_pdfsb0f6f69d619725c8 = _data_pdfs + _stride_pdfs_1 * y + _stride_pdfs_1 * neighbour_offset_y[dir] + _stride_pdfs_2 * z + _stride_pdfs_2 * neighbour_offset_z[dir] + _stride_pdfs_3 * f_in_inv_dir_idx[dir];
    uint8_t *RESTRICT _data_indexVector_116 = _data_indexVector + 16;
    uint8_t *RESTRICT _data_indexVector_120 = _data_indexVector + 20;
    uint8_t *RESTRICT _data_indexVector_124 = _data_indexVector + 24;
    float *RESTRICT _data_pdfs_10_20b9bbe59f808ba907 = _data_pdfs + _stride_pdfs_1 * y + _stride_pdfs_2 * z + _stride_pdfs_3 * dir;
    _data_pdfsb0f6f69d619725c8[_stride_pdfs_0 * x + _stride_pdfs_0 * neighbour_offset_x[dir]] = -rho * (6.0f * ((float)(neighbour_offset_x[dir])) * *((float *)(&_data_indexVector_116[28 * blockDim.x * blockIdx.x + 28 * threadIdx.x])) + 6.0f * ((float)(neighbour_offset_y[dir])) * *((float *)(&_data_indexVector_120[28 * blockDim.x * blockIdx.x + 28 * threadIdx.x])) + 6.0f * ((float)(neighbour_offset_z[dir])) * *((float *)(&_data_indexVector_124[28 * blockDim.x * blockIdx.x + 28 * threadIdx.x]))) * weights[dir] + _data_pdfs_10_20b9bbe59f808ba907[_stride_pdfs_0 * x];
  }
}
} // namespace internal_dynamicubbsingleprecisioncuda_boundary_DynamicUBBSinglePrecisionCUDA

// NOLINTEND(readability-non-const-parameter*)

#if defined(__NVCC__)
#if defined(__NVCC_DIAG_PRAGMA_SUPPORT__)
#pragma nv_diagnostic pop
#else
#pragma pop
#endif // defined(__NVCC_DIAG_PRAGMA_SUPPORT__)
#elif defined(__clang__)
#if defined(__CUDA__)
#if defined(__CUDA_ARCH__)
// clang compiling CUDA code in device mode
#pragma clang diagnostic pop
#else
// clang compiling CUDA code in host mode
#pragma clang diagnostic pop
#endif // defined(__CUDA_ARCH__)
#endif // defined(__CUDA__)
#elif defined(__GNUC__) or defined(__GNUG__)
#pragma GCC diagnostic pop
#endif

void DynamicUBBSinglePrecisionCUDA::run_impl(IBlock *block, IndexVectors::Type type, gpuStream_t stream) {
  auto *indexVectors = block->getData<IndexVectors>(indexVectorID);
  int32_t indexVectorSize = int32_c(indexVectors->indexVector(type).size());
  if (indexVectorSize == 0)
    return;

  auto pointer = indexVectors->pointerGpu(type);

  uint8_t *_data_indexVector = reinterpret_cast<uint8_t *>(pointer);

  auto pdfs = block->getData<gpu::GPUField<float>>(pdfsID);

  WALBERLA_ASSERT_GREATER_EQUAL(0, -int_c(pdfs->nrOfGhostLayers()))
  float *RESTRICT _data_pdfs = pdfs->dataAt(0, 0, 0, 0);
  const int64_t _stride_pdfs_0 = int64_t(pdfs->xStride());
  const int64_t _stride_pdfs_1 = int64_t(pdfs->yStride());
  const int64_t _stride_pdfs_2 = int64_t(pdfs->zStride());
  const int64_t _stride_pdfs_3 = int64_t(1 * int64_t(pdfs->fStride()));
  dim3 _block(uint32_c(((256 < indexVectorSize) ? 256 : indexVectorSize)), uint32_c(1), uint32_c(1));
  dim3 _grid(uint32_c(((indexVectorSize) % (((256 < indexVectorSize) ? 256 : indexVectorSize)) == 0 ? (int64_t)(indexVectorSize) / (int64_t)(((256 < indexVectorSize) ? 256 : indexVectorSize)) : ((int64_t)(indexVectorSize) / (int64_t)(((256 < indexVectorSize) ? 256 : indexVectorSize))) + 1)), uint32_c(1), uint32_c(1));
  internal_dynamicubbsingleprecisioncuda_boundary_DynamicUBBSinglePrecisionCUDA::dynamicubbsingleprecisioncuda_boundary_DynamicUBBSinglePrecisionCUDA<<<_grid, _block, 0, stream>>>(_data_indexVector, _data_pdfs, _stride_pdfs_0, _stride_pdfs_1, _stride_pdfs_2, _stride_pdfs_3, indexVectorSize);
}

void DynamicUBBSinglePrecisionCUDA::run(IBlock *block, gpuStream_t stream) {
  run_impl(block, IndexVectors::ALL, stream);
}

void DynamicUBBSinglePrecisionCUDA::inner(IBlock *block, gpuStream_t stream) {
  run_impl(block, IndexVectors::INNER, stream);
}

void DynamicUBBSinglePrecisionCUDA::outer(IBlock *block, gpuStream_t stream) {
  run_impl(block, IndexVectors::OUTER, stream);
}

} // namespace lbm
} // namespace walberla
