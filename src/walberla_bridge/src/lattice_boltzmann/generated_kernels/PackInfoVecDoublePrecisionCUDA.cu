#include "hip/hip_runtime.h"
//======================================================================================================================
//
//  This file is part of waLBerla. waLBerla is free software: you can
//  redistribute it and/or modify it under the terms of the GNU General Public
//  License as published by the Free Software Foundation, either version 3 of
//  the License, or (at your option) any later version.
//
//  waLBerla is distributed in the hope that it will be useful, but WITHOUT
//  ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or
//  FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public License
//  for more details.
//
//  You should have received a copy of the GNU General Public License along
//  with waLBerla (see COPYING.txt). If not, see <http://www.gnu.org/licenses/>.
//
//! \\file PackInfoVecDoublePrecisionCUDA.cpp
//! \\author pystencils
//======================================================================================================================

// kernel generated with pystencils v1.3.7, lbmpy v1.3.7, sympy v1.12.1, lbmpy_walberla/pystencils_walberla from waLBerla commit f36fa0a68bae59f0b516f6587ea8fa7c24a41141

#include "core/DataTypes.h"
#include "core/cell/CellInterval.h"

#include "domain_decomposition/IBlock.h"

#include "stencil/Directions.h"

#include "core/DataTypes.h"
#include "core/cell/CellInterval.h"
#include "domain_decomposition/IBlock.h"
#include "stencil/Directions.h"

#include "PackInfoVecDoublePrecisionCUDA.h"

#define FUNC_PREFIX __global__

#if defined(__NVCC__)
#define RESTRICT __restrict__
#if defined(__NVCC_DIAG_PRAGMA_SUPPORT__)
#pragma nv_diagnostic push
#pragma nv_diag_suppress 177 // unused variable
#else
#pragma push
#pragma diag_suppress 177 // unused variable
#endif                    // defined(__NVCC_DIAG_PRAGMA_SUPPORT__)
#elif defined(__clang__)
#if defined(__CUDA__)
#if defined(__CUDA_ARCH__)
// clang compiling CUDA code in device mode
#define RESTRICT __restrict__
#pragma clang diagnostic push
#pragma clang diagnostic ignored "-Wunused-variable"
#else
// clang compiling CUDA code in host mode
#define RESTRICT __restrict__
#pragma clang diagnostic push
#pragma clang diagnostic ignored "-Wunused-variable"
#endif // defined(__CUDA_ARCH__)
#endif // defined(__CUDA__)
#elif defined(__GNUC__) or defined(__GNUG__)
#define RESTRICT __restrict__
#pragma GCC diagnostic push
#pragma GCC diagnostic ignored "-Wunused-variable"
#elif defined(_MSC_VER)
#define RESTRICT __restrict
#else
#define RESTRICT
#endif

namespace walberla {
namespace pystencils {

using walberla::cell::CellInterval;
using walberla::stencil::Direction;

namespace internal_pack_SW_BW_W_TW_NW_BS_S_TS_B_C_T_BN_N_TN_SE_BE_E_TE_NE {
static FUNC_PREFIX __launch_bounds__(256) void pack_SW_BW_W_TW_NW_BS_S_TS_B_C_T_BN_N_TN_SE_BE_E_TE_NE(double *RESTRICT _data_buffer, double *RESTRICT const _data_field, int64_t const _size_field_0, int64_t const _size_field_1, int64_t const _size_field_2, int64_t const _stride_field_0, int64_t const _stride_field_1, int64_t const _stride_field_2, int64_t const _stride_field_3) {
  if (blockDim.x * blockIdx.x + threadIdx.x < _size_field_0 && blockDim.y * blockIdx.y + threadIdx.y < _size_field_1 && blockDim.z * blockIdx.z + threadIdx.z < _size_field_2) {
    const int64_t ctr_0 = blockDim.x * blockIdx.x + threadIdx.x;
    const int64_t ctr_1 = blockDim.y * blockIdx.y + threadIdx.y;
    const int64_t ctr_2 = blockDim.z * blockIdx.z + threadIdx.z;
    _data_buffer[3 * _size_field_0 * _size_field_1 * ctr_2 + 3 * _size_field_0 * ctr_1 + 3 * ctr_0] = _data_field[_stride_field_0 * ctr_0 + _stride_field_1 * ctr_1 + _stride_field_2 * ctr_2];
    _data_buffer[3 * _size_field_0 * _size_field_1 * ctr_2 + 3 * _size_field_0 * ctr_1 + 3 * ctr_0 + 1] = _data_field[_stride_field_0 * ctr_0 + _stride_field_1 * ctr_1 + _stride_field_2 * ctr_2 + _stride_field_3];
    _data_buffer[3 * _size_field_0 * _size_field_1 * ctr_2 + 3 * _size_field_0 * ctr_1 + 3 * ctr_0 + 2] = _data_field[_stride_field_0 * ctr_0 + _stride_field_1 * ctr_1 + _stride_field_2 * ctr_2 + 2 * _stride_field_3];
  }
}
} // namespace internal_pack_SW_BW_W_TW_NW_BS_S_TS_B_C_T_BN_N_TN_SE_BE_E_TE_NE

namespace internal_unpack_SW_BW_W_TW_NW_BS_S_TS_B_C_T_BN_N_TN_SE_BE_E_TE_NE {
static FUNC_PREFIX __launch_bounds__(256) void unpack_SW_BW_W_TW_NW_BS_S_TS_B_C_T_BN_N_TN_SE_BE_E_TE_NE(double *RESTRICT const _data_buffer, double *RESTRICT _data_field, int64_t const _size_field_0, int64_t const _size_field_1, int64_t const _size_field_2, int64_t const _stride_field_0, int64_t const _stride_field_1, int64_t const _stride_field_2, int64_t const _stride_field_3) {
  if (blockDim.x * blockIdx.x + threadIdx.x < _size_field_0 && blockDim.y * blockIdx.y + threadIdx.y < _size_field_1 && blockDim.z * blockIdx.z + threadIdx.z < _size_field_2) {
    const int64_t ctr_0 = blockDim.x * blockIdx.x + threadIdx.x;
    const int64_t ctr_1 = blockDim.y * blockIdx.y + threadIdx.y;
    const int64_t ctr_2 = blockDim.z * blockIdx.z + threadIdx.z;
    _data_field[_stride_field_0 * ctr_0 + _stride_field_1 * ctr_1 + _stride_field_2 * ctr_2] = _data_buffer[3 * _size_field_0 * _size_field_1 * ctr_2 + 3 * _size_field_0 * ctr_1 + 3 * ctr_0];
    _data_field[_stride_field_0 * ctr_0 + _stride_field_1 * ctr_1 + _stride_field_2 * ctr_2 + _stride_field_3] = _data_buffer[3 * _size_field_0 * _size_field_1 * ctr_2 + 3 * _size_field_0 * ctr_1 + 3 * ctr_0 + 1];
    _data_field[_stride_field_0 * ctr_0 + _stride_field_1 * ctr_1 + _stride_field_2 * ctr_2 + 2 * _stride_field_3] = _data_buffer[3 * _size_field_0 * _size_field_1 * ctr_2 + 3 * _size_field_0 * ctr_1 + 3 * ctr_0 + 2];
  }
}
} // namespace internal_unpack_SW_BW_W_TW_NW_BS_S_TS_B_C_T_BN_N_TN_SE_BE_E_TE_NE

void PackInfoVecDoublePrecisionCUDA::pack(Direction dir, unsigned char *byte_buffer, IBlock *block, gpuStream_t stream) {
  double *buffer = reinterpret_cast<double *>(byte_buffer);

  auto field = block->getData<gpu::GPUField<double>>(fieldID);

  CellInterval ci;
  field->getSliceBeforeGhostLayer(dir, ci, 1, false);

  switch (dir) {
  case stencil::SW:
  case stencil::BW:
  case stencil::W:
  case stencil::TW:
  case stencil::NW:
  case stencil::BS:
  case stencil::S:
  case stencil::TS:
  case stencil::B:
  case stencil::C:
  case stencil::T:
  case stencil::BN:
  case stencil::N:
  case stencil::TN:
  case stencil::SE:
  case stencil::BE:
  case stencil::E:
  case stencil::TE:
  case stencil::NE: {
    double *RESTRICT _data_buffer = buffer;
    WALBERLA_ASSERT_GREATER_EQUAL(ci.xMin(), -int_c(field->nrOfGhostLayers()))
    WALBERLA_ASSERT_GREATER_EQUAL(ci.yMin(), -int_c(field->nrOfGhostLayers()))
    WALBERLA_ASSERT_GREATER_EQUAL(ci.zMin(), -int_c(field->nrOfGhostLayers()))
    double *RESTRICT const _data_field = field->dataAt(ci.xMin(), ci.yMin(), ci.zMin(), 0);
    WALBERLA_ASSERT_GREATER_EQUAL(field->xSizeWithGhostLayer(), int64_t(int64_c(ci.xSize()) + 0))
    const int64_t _size_field_0 = int64_t(int64_c(ci.xSize()) + 0);
    WALBERLA_ASSERT_GREATER_EQUAL(field->ySizeWithGhostLayer(), int64_t(int64_c(ci.ySize()) + 0))
    const int64_t _size_field_1 = int64_t(int64_c(ci.ySize()) + 0);
    WALBERLA_ASSERT_GREATER_EQUAL(field->zSizeWithGhostLayer(), int64_t(int64_c(ci.zSize()) + 0))
    const int64_t _size_field_2 = int64_t(int64_c(ci.zSize()) + 0);
    const int64_t _stride_field_0 = int64_t(field->xStride());
    const int64_t _stride_field_1 = int64_t(field->yStride());
    const int64_t _stride_field_2 = int64_t(field->zStride());
    const int64_t _stride_field_3 = int64_t(1 * int64_t(field->fStride()));
    dim3 _block(uint32_c(((128 < _size_field_0) ? 128 : _size_field_0)), uint32_c(((1024 < ((_size_field_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0)))) ? _size_field_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0))))) ? 1024 : ((_size_field_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0)))) ? _size_field_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0)))))), uint32_c(((64 < ((_size_field_2 < ((int64_t)(256) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0) * ((_size_field_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0)))) ? _size_field_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0))))))) ? _size_field_2 : ((int64_t)(256) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0) * ((_size_field_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0)))) ? _size_field_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0)))))))) ? 64 : ((_size_field_2 < ((int64_t)(256) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0) * ((_size_field_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0)))) ? _size_field_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0))))))) ? _size_field_2 : ((int64_t)(256) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0) * ((_size_field_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0)))) ? _size_field_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0))))))))));
    dim3 _grid(uint32_c(((_size_field_0) % (((128 < _size_field_0) ? 128 : _size_field_0)) == 0 ? (int64_t)(_size_field_0) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0)) : ((int64_t)(_size_field_0) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0))) + 1)), uint32_c(((_size_field_1) % (((1024 < ((_size_field_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0)))) ? _size_field_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0))))) ? 1024 : ((_size_field_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0)))) ? _size_field_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0)))))) == 0 ? (int64_t)(_size_field_1) / (int64_t)(((1024 < ((_size_field_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0)))) ? _size_field_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0))))) ? 1024 : ((_size_field_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0)))) ? _size_field_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0)))))) : ((int64_t)(_size_field_1) / (int64_t)(((1024 < ((_size_field_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0)))) ? _size_field_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0))))) ? 1024 : ((_size_field_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0)))) ? _size_field_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0))))))) + 1)), uint32_c(((_size_field_2) % (((64 < ((_size_field_2 < ((int64_t)(256) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0) * ((_size_field_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0)))) ? _size_field_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0))))))) ? _size_field_2 : ((int64_t)(256) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0) * ((_size_field_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0)))) ? _size_field_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0)))))))) ? 64 : ((_size_field_2 < ((int64_t)(256) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0) * ((_size_field_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0)))) ? _size_field_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0))))))) ? _size_field_2 : ((int64_t)(256) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0) * ((_size_field_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0)))) ? _size_field_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0))))))))) == 0 ? (int64_t)(_size_field_2) / (int64_t)(((64 < ((_size_field_2 < ((int64_t)(256) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0) * ((_size_field_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0)))) ? _size_field_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0))))))) ? _size_field_2 : ((int64_t)(256) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0) * ((_size_field_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0)))) ? _size_field_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0)))))))) ? 64 : ((_size_field_2 < ((int64_t)(256) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0) * ((_size_field_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0)))) ? _size_field_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0))))))) ? _size_field_2 : ((int64_t)(256) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0) * ((_size_field_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0)))) ? _size_field_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0))))))))) : ((int64_t)(_size_field_2) / (int64_t)(((64 < ((_size_field_2 < ((int64_t)(256) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0) * ((_size_field_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0)))) ? _size_field_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0))))))) ? _size_field_2 : ((int64_t)(256) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0) * ((_size_field_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0)))) ? _size_field_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0)))))))) ? 64 : ((_size_field_2 < ((int64_t)(256) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0) * ((_size_field_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0)))) ? _size_field_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0))))))) ? _size_field_2 : ((int64_t)(256) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0) * ((_size_field_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0)))) ? _size_field_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0)))))))))) + 1)));
    internal_pack_SW_BW_W_TW_NW_BS_S_TS_B_C_T_BN_N_TN_SE_BE_E_TE_NE::pack_SW_BW_W_TW_NW_BS_S_TS_B_C_T_BN_N_TN_SE_BE_E_TE_NE<<<_grid, _block, 0, stream>>>(_data_buffer, _data_field, _size_field_0, _size_field_1, _size_field_2, _stride_field_0, _stride_field_1, _stride_field_2, _stride_field_3);
    break;
  }

  default:
    return;
  }
}

void PackInfoVecDoublePrecisionCUDA::unpack(Direction dir, unsigned char *byte_buffer, IBlock *block, gpuStream_t stream) {
  double *buffer = reinterpret_cast<double *>(byte_buffer);

  auto field = block->getData<gpu::GPUField<double>>(fieldID);

  CellInterval ci;
  field->getGhostRegion(dir, ci, 1, false);
  auto communciationDirection = stencil::inverseDir[dir];

  switch (communciationDirection) {
  case stencil::SW:
  case stencil::BW:
  case stencil::W:
  case stencil::TW:
  case stencil::NW:
  case stencil::BS:
  case stencil::S:
  case stencil::TS:
  case stencil::B:
  case stencil::C:
  case stencil::T:
  case stencil::BN:
  case stencil::N:
  case stencil::TN:
  case stencil::SE:
  case stencil::BE:
  case stencil::E:
  case stencil::TE:
  case stencil::NE: {
    double *RESTRICT const _data_buffer = buffer;
    WALBERLA_ASSERT_GREATER_EQUAL(ci.xMin(), -int_c(field->nrOfGhostLayers()))
    WALBERLA_ASSERT_GREATER_EQUAL(ci.yMin(), -int_c(field->nrOfGhostLayers()))
    WALBERLA_ASSERT_GREATER_EQUAL(ci.zMin(), -int_c(field->nrOfGhostLayers()))
    double *RESTRICT _data_field = field->dataAt(ci.xMin(), ci.yMin(), ci.zMin(), 0);
    WALBERLA_ASSERT_GREATER_EQUAL(field->xSizeWithGhostLayer(), int64_t(int64_c(ci.xSize()) + 0))
    const int64_t _size_field_0 = int64_t(int64_c(ci.xSize()) + 0);
    WALBERLA_ASSERT_GREATER_EQUAL(field->ySizeWithGhostLayer(), int64_t(int64_c(ci.ySize()) + 0))
    const int64_t _size_field_1 = int64_t(int64_c(ci.ySize()) + 0);
    WALBERLA_ASSERT_GREATER_EQUAL(field->zSizeWithGhostLayer(), int64_t(int64_c(ci.zSize()) + 0))
    const int64_t _size_field_2 = int64_t(int64_c(ci.zSize()) + 0);
    const int64_t _stride_field_0 = int64_t(field->xStride());
    const int64_t _stride_field_1 = int64_t(field->yStride());
    const int64_t _stride_field_2 = int64_t(field->zStride());
    const int64_t _stride_field_3 = int64_t(1 * int64_t(field->fStride()));
    dim3 _block(uint32_c(((128 < _size_field_0) ? 128 : _size_field_0)), uint32_c(((1024 < ((_size_field_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0)))) ? _size_field_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0))))) ? 1024 : ((_size_field_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0)))) ? _size_field_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0)))))), uint32_c(((64 < ((_size_field_2 < ((int64_t)(256) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0) * ((_size_field_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0)))) ? _size_field_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0))))))) ? _size_field_2 : ((int64_t)(256) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0) * ((_size_field_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0)))) ? _size_field_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0)))))))) ? 64 : ((_size_field_2 < ((int64_t)(256) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0) * ((_size_field_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0)))) ? _size_field_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0))))))) ? _size_field_2 : ((int64_t)(256) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0) * ((_size_field_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0)))) ? _size_field_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0))))))))));
    dim3 _grid(uint32_c(((_size_field_0) % (((128 < _size_field_0) ? 128 : _size_field_0)) == 0 ? (int64_t)(_size_field_0) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0)) : ((int64_t)(_size_field_0) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0))) + 1)), uint32_c(((_size_field_1) % (((1024 < ((_size_field_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0)))) ? _size_field_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0))))) ? 1024 : ((_size_field_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0)))) ? _size_field_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0)))))) == 0 ? (int64_t)(_size_field_1) / (int64_t)(((1024 < ((_size_field_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0)))) ? _size_field_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0))))) ? 1024 : ((_size_field_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0)))) ? _size_field_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0)))))) : ((int64_t)(_size_field_1) / (int64_t)(((1024 < ((_size_field_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0)))) ? _size_field_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0))))) ? 1024 : ((_size_field_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0)))) ? _size_field_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0))))))) + 1)), uint32_c(((_size_field_2) % (((64 < ((_size_field_2 < ((int64_t)(256) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0) * ((_size_field_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0)))) ? _size_field_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0))))))) ? _size_field_2 : ((int64_t)(256) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0) * ((_size_field_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0)))) ? _size_field_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0)))))))) ? 64 : ((_size_field_2 < ((int64_t)(256) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0) * ((_size_field_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0)))) ? _size_field_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0))))))) ? _size_field_2 : ((int64_t)(256) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0) * ((_size_field_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0)))) ? _size_field_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0))))))))) == 0 ? (int64_t)(_size_field_2) / (int64_t)(((64 < ((_size_field_2 < ((int64_t)(256) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0) * ((_size_field_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0)))) ? _size_field_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0))))))) ? _size_field_2 : ((int64_t)(256) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0) * ((_size_field_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0)))) ? _size_field_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0)))))))) ? 64 : ((_size_field_2 < ((int64_t)(256) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0) * ((_size_field_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0)))) ? _size_field_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0))))))) ? _size_field_2 : ((int64_t)(256) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0) * ((_size_field_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0)))) ? _size_field_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0))))))))) : ((int64_t)(_size_field_2) / (int64_t)(((64 < ((_size_field_2 < ((int64_t)(256) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0) * ((_size_field_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0)))) ? _size_field_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0))))))) ? _size_field_2 : ((int64_t)(256) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0) * ((_size_field_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0)))) ? _size_field_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0)))))))) ? 64 : ((_size_field_2 < ((int64_t)(256) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0) * ((_size_field_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0)))) ? _size_field_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0))))))) ? _size_field_2 : ((int64_t)(256) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0) * ((_size_field_1 < 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0)))) ? _size_field_1 : 2 * ((int64_t)(128) / (int64_t)(((128 < _size_field_0) ? 128 : _size_field_0)))))))))) + 1)));
    internal_unpack_SW_BW_W_TW_NW_BS_S_TS_B_C_T_BN_N_TN_SE_BE_E_TE_NE::unpack_SW_BW_W_TW_NW_BS_S_TS_B_C_T_BN_N_TN_SE_BE_E_TE_NE<<<_grid, _block, 0, stream>>>(_data_buffer, _data_field, _size_field_0, _size_field_1, _size_field_2, _stride_field_0, _stride_field_1, _stride_field_2, _stride_field_3);
    break;
  }

  default:
    return;
  }
}

uint_t PackInfoVecDoublePrecisionCUDA::size(stencil::Direction dir, IBlock *block) {
  auto field = block->getData<gpu::GPUField<double>>(fieldID);

  CellInterval ci;
  field->getGhostRegion(dir, ci, 1, false);

  uint_t elementsPerCell = 0;

  switch (dir) {
  case stencil::SW:
  case stencil::BW:
  case stencil::W:
  case stencil::TW:
  case stencil::NW:
  case stencil::BS:
  case stencil::S:
  case stencil::TS:
  case stencil::B:
  case stencil::C:
  case stencil::T:
  case stencil::BN:
  case stencil::N:
  case stencil::TN:
  case stencil::SE:
  case stencil::BE:
  case stencil::E:
  case stencil::TE:
  case stencil::NE:
    elementsPerCell = 3;
    break;

  default:
    elementsPerCell = 0;
  }
  return ci.numCells() * elementsPerCell * sizeof(double);
}

} // namespace pystencils
} // namespace walberla
